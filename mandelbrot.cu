#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "panelButton.h"
#include "text.h"
#include "parameters.h"
#include "color.h"

__global__ void fractal_kernel(int* d_pixels, int width, int height, double x_min, double x_max, double y_min, double y_max, double CX, double CY, bool MANDELBROT) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    double real = x_min + x * (x_max - x_min) / width;
    double imag = y_min + y * (y_max - y_min) / height;

    double z_real = real;
    double z_imag = imag;

    /*
        mandelbrot: z� + real + imag
        z� - 1
        z� -0.2 + 0.7i
    */
    if (MANDELBROT) {
        CX = real;
        CY = imag;
    }

    double cx = CX;
    double cy = CY;
    int value = 0;
    double tmp_real = z_real;
    double tmp_imag = z_imag;
    for (int i = 0; i < MAX_ITERATIONS; ++i) {
        tmp_real = z_real;
        tmp_imag = z_imag;
        if (tmp_real * tmp_real + tmp_imag * tmp_imag > 4.0f) {
            value = i;
            break;
        }
        z_real = tmp_real * tmp_real - tmp_imag * tmp_imag + cx;
        z_imag = 2 * tmp_real * tmp_imag + cy; // 2ab + b� avec (x*i)�
    }

    d_pixels[y * width + x] = value;
}

void switchMandelbrot(sf::RenderWindow &window, bool &MANDELBROT, PanelButtons &panelX, PanelButtons &panelY, Button &mandelbrotButton)
{
    if (sf::Mouse::isButtonPressed(sf::Mouse::Button::Left) && mandelbrotButton.isClicked(sf::Mouse::getPosition(window))) {
        MANDELBROT = !MANDELBROT;
        if (MANDELBROT) {
            panelX.addButton.shape.setFillColor(sf::Color(128, 128, 128));
            panelX.minButton.shape.setFillColor(sf::Color(128, 128, 128));
            panelY.addButton.shape.setFillColor(sf::Color(128, 128, 128));
            panelY.minButton.shape.setFillColor(sf::Color(128, 128, 128));
        }
        else {
            panelX.minButton.shape.setFillColor(sf::Color::White);
            panelY.addButton.shape.setFillColor(sf::Color::White);
            panelX.addButton.shape.setFillColor(sf::Color::White);
            panelY.minButton.shape.setFillColor(sf::Color::White);
        }
    }
    else if (!sf::Mouse::isButtonPressed(sf::Mouse::Button::Left) && mandelbrotButton.isClicked(sf::Mouse::getPosition(window))) {
        mandelbrotButton.shape.setFillColor(sf::Color::White);
    }
}

int main() {
    bool MANDELBROT = false;

    int width = WIDTH;
    int height = HEIGHT;

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
 
    sf::Image icon;
    icon.loadFromFile("window_icon.jpg");
    // Set up the window
    sf::RenderWindow window(sf::VideoMode(width, height), "Mandelbrot Fractal");
    window.setFramerateLimit(30);
    window.setIcon(icon.getSize().x, icon.getSize().y, icon.getPixelsPtr());

    // Set up the image
    sf::Image image;
    image.create(width, height, sf::Color::Black);
    sf::Texture texture;
    texture.loadFromImage(image);
    sf::Sprite sprite;
    sprite.setTexture(texture);

    // Set up the CUDA device and arrays
    int* d_pixels;
    hipMalloc((void**)&d_pixels, width * height * sizeof(int));

    double x_min = -2.0f;
    double x_max = 1.0f;
    double y_min = -1.0f;
    double y_max = 1.0f;

    // Set up the zoom factor
    double zoom_factor = 1.0f;

    // Set up the mouse positions
    int mouse_x = 0;
    int mouse_y = 0;
    sf::Clock clock;
    bool z = true;
    bool hideButtons = false;

    PanelButtons panelX(sf::Vector2f(20, 20), sf::Vector2f(50, 20), 10, std::vector<sf::Vector2f>{sf::Vector2f(25, 8), sf::Vector2f(51, 10)}, "CX");
    PanelButtons panelY(sf::Vector2f(20, 60), sf::Vector2f(50, 60), 10, std::vector<sf::Vector2f>{sf::Vector2f(25, 48), sf::Vector2f(51, 50)}, "CY");
    PanelButtons panelStep(sf::Vector2f(20, 870), sf::Vector2f(50, 870), 10, std::vector<sf::Vector2f>{sf::Vector2f(25, 858), sf::Vector2f(51, 860)}, "Step : 0.010");
    Button mandelbrotButton(sf::Vector2f(20, 100), 10, "Mandelbrot", sf::Vector2f(50, 100));
    double CX = -0.2f;
    double CY = 0.7f;
    // Run the main loop
    while (window.isOpen()) {
        sf::Event event;
        while (window.pollEvent(event)) {
            switchMandelbrot(window, MANDELBROT, panelX, panelY, mandelbrotButton);
            updateConstants(window, MANDELBROT, panelX, panelY, CX, CY, panelStep.step);
            updateStep(window, panelStep);

            if (event.type == sf::Event::KeyPressed && event.key.code == sf::Keyboard::H) {
                hideButtons = !hideButtons;
            }

            if (event.type == sf::Event::Closed || sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {
                window.close();
            }
            if (event.type == sf::Event::MouseWheelScrolled) {
                // Calculate the new zoom level
                if (z) {
                    z = false;
                    zoom_factor *= 1.0f + event.mouseWheelScroll.delta / 10.0f;
                }
                else {
                    z = true;
                    zoom_factor = 1.0;
                }

                double x_center = x_min + (x_max - x_min) * mouse_x / width;
                double y_center = y_min + (y_max - y_min) * mouse_y / height;
                x_min = x_center - (x_center - x_min) * zoom_factor;
                x_max = x_center + (x_max - x_center) * zoom_factor;
                y_min = y_center - (y_center - y_min) * zoom_factor;
                y_max = y_center + (y_max - y_center) * zoom_factor;
            }
            if (event.type == sf::Event::MouseMoved) {
                mouse_x = event.mouseMove.x;
                mouse_y = event.mouseMove.y;
            }
        }

        // Launch the kernel
        fractal_kernel << <grid, block >> > (d_pixels, width, height, x_min, x_max, y_min, y_max, CX, CY, MANDELBROT);
        // Copy the results back to the host
        int *pixels = new int[width * height];
        hipMemcpy(pixels, d_pixels, width * height * sizeof(int), hipMemcpyDeviceToHost);

        if (clock.getElapsedTime().asMilliseconds() > 50) {
            render_color(&image, pixels, width, height);
            texture.loadFromImage(image);
            sprite.setTexture(texture);
            // Draw the image
            window.clear();
            window.draw(sprite);
            if (!hideButtons) {
                panelX.draw(window);
                panelY.draw(window);
                panelStep.draw(window);
                mandelbrotButton.draw(window);
                print_equation(CX, CY, MANDELBROT, window);
            }
            window.display();
            clock.restart();
        }
        // Clean up
        delete[] pixels;
    }

    hipFree(d_pixels);

    return 0;
}